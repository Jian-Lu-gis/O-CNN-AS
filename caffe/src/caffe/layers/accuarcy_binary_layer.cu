#include "hip/hip_runtime.h"
#include "caffe/layers/accuracy_binary_layer.hpp"
#include "caffe/util/math_functions.hpp"

#include <thrust/count.h>
#include <thrust/execution_policy.h>

namespace caffe {

template <typename Dtype>
__global__ void accuracy_forward_kernel(int* buffer, const Dtype* bottom_data,
    const Dtype* bottom_label, const int n) {
  CUDA_KERNEL_LOOP(i, n) {
    int label_gt = static_cast<int>(bottom_label[i]);
    int label = static_cast<int>(bottom_data[i] > 0);

    buffer[i] = label_gt * 2 + label;
  }
}

template <typename Dtype>
void AccuracyBinaryLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_label = bottom[1]->gpu_data();
  buffer_.Reshape(bottom[0]->shape());
  int* buffer = buffer_.mutable_gpu_data();
  int count = bottom[0]->count();
  accuracy_forward_kernel <Dtype> <<< CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >>> (
      buffer_.mutable_gpu_data(), bottom_data, bottom_label, count);

  int num[4] = { 0 };
  for (int i = 0; i < 4; ++i) {
    num[i] = thrust::count(thrust::device, buffer, buffer + count, i);
  }

  top[0]->mutable_cpu_data()[0] = Dtype(num[0] + num[3]) / Dtype(count);
  if (top.size() == 2) {
    Dtype* top_data = top[1]->mutable_cpu_data();
    for (int i = 0; i < 4; ++i) {
      top_data[i] = Dtype(num[i]) / Dtype(count);
    }
  }
}

template <typename Dtype>
void AccuracyBinaryLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < propagate_down.size(); ++i) {
    if (propagate_down[i]) NOT_IMPLEMENTED;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AccuracyBinaryLayer);

}  // namespace caffe
